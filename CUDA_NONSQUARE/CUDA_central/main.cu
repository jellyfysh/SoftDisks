#include "hip/hip_runtime.h"
//
//   Copyright (c) 2023, Yoshihiko Nishikawa, Werner Krauth, and A. C. Maggs
//
//   CUDA code for massively parallelized Monte Carlo simulation of
//   two-dimensional disks
//
//   URL: https://github.com/jellyfysh/SoftDisks
//   See LICENSE for copyright information
//
//   If you use this code or find it useful, please cite the following paper:
//
//   @article{PhysRevE.108.024103,
//       title = {Liquid-hexatic transition for soft disks},
//       author = {Nishikawa, Yoshihiko and Krauth, Werner and Maggs, A. C.},
//       journal = {Phys. Rev. E},
//       volume = {108},
//       issue = {2},
//       pages = {024103},
//       numpages = {7},
//       year = {2023},
//       month = {Aug},
//       publisher = {American Physical Society},
//       doi = {10.1103/PhysRevE.108.024103},
//       url = {https://link.aps.org/doi/10.1103/PhysRevE.108.024103}
//   }
//
//
#include "Soft.h"
#include "output_data.h"
#include <fstream>
//#include <hip/hip_runtime_api.h>
#include "timer.h"
#include <sstream> 
#include <iomanip>
#include <iostream> 
int freq_output = -1;
int freq_energy = 50;


float density;
double Lbox, Lboy;
double Lblockx, Lblocky;
float *rx, *ry;
short *nparticle;
float *devrx, *devry;
short *devnparticle;
float rshiftx = 0, rshifty = 0;
float BETA;

// Texture memory
texture<float, 1, hipReadModeElementType> read_rx;
texture<float, 1, hipReadModeElementType> read_ry;
texture<short, 1, hipReadModeElementType> read_nparticle;

int main(int argc, char **argv){
  Timer mytime;
#ifdef INPUT_DENSITY

  if(argc != 3){
    cerr << "Input density and BETA" << endl;
    assert(argc == 3);
  } else {
    density = atof(argv[1]);
    BETA = atof(argv[2]);
  }

  Lbox = sqrt(2.0 * (double)NNN / (double)density / sqrt(3.0));
  Lboy = Lbox * sqrt(3.0) * 0.5;
  Lblockx = (float)(Lbox / (double)Nblock);
  Lblocky = (float)(Lboy / (double)Nblock);

  if(Lblockx < cutoff || Lblocky < cutoff){
    cerr << "Error, Lblockx " << Lblockx << "and Lblocky " << Lblocky << " must be larger than cutoff " << cutoff << endl;
    assert(Lblockx > cutoff && Lblocky > cutoff);
  }

#endif
  
  cerr.precision(12);
  cerr << " N = " << NNN 
       << ", density = " << density
       << ", BETA = " << BETA
       << ", Lbox = " << Lbox
       << ", Lboy = " << Lboy
       << ", Lblock = " << Lblockx
       << ", Lblocky = " << Lblocky << endl;
  cerr << "Lbox\t" << Lbox << " Lboy\t" << Lboy << endl;
  //hipProfilerStart();
  int num_gpus, gpu_id;
  hipGetDeviceCount(&num_gpus);
  hipGetDevice(&gpu_id);

  if(1){
    cout<<"Nrow\t"<<Nrow<<endl;
    cout<<"NNN\t"<<NNN<<endl;
    cout<<"rpotential\t"<<rpotential<<endl;
    cout<<"MaxMCS\t"<<MaxMCS<<endl;
    cout<<"cutoff\t"<<cutoff<<endl;
    cout<<"Nblock\t"<<Nblock<<endl;
    cout<<"nmax\t"<<nmax<<endl;
    cout<<"BLOCKL\t"<<BLOCKL<<endl;
    cout<<"GRIDL\t"<<GRIDL<<endl;
    cout<<"BLOCKS\t"<<BLOCKS<<endl;
    cout<<"THREADS\t"<<THREADS<<endl;
    cout<<"SHIFT_BLOCKL\t"<<endl;
    cout<<"SHIFT_GRIDL\t"<<SHIFT_GRIDL<<endl;
    cout<<"MBLOCKL\t"<<MBLOCKL<<endl;
    cout<<"MGRIDL\t"<<MGRIDL<<endl;
    cout<<"NCURAND\t"<<NCURAND<<endl<<endl;
  }

  {// Allocate memory on gpu
    hipMallocManaged(&rx, Nblock * Nblock * nmax * sizeof(float));
    hipMallocManaged(&ry, Nblock * Nblock * nmax * sizeof(float));
    hipMallocManaged(&nparticle, Nblock * Nblock * sizeof(short));
    hipMallocManaged(&devrx, Nblock * Nblock * nmax * sizeof(float));
    hipMallocManaged(&devry, Nblock * Nblock * nmax * sizeof(float));
    hipMallocManaged(&devnparticle, Nblock * Nblock * sizeof(short));
  }


  int seed = (int)(density * 100 + NNN) + time(0) ;
  Set_Mersenne_Twister_GPU(seed);
  hipDeviceSynchronize();

  
  cout.precision(7);
  cout.setf(ios::scientific);

  freq_output = (MaxMCS < N_config ? MaxMCS : (int)(MaxMCS / N_config));
  if(rpotential < 0)
    freq_energy = 100;

  
  Set_init_conf();
  unsigned int iter = 0;
  ostringstream ss ;
  ss << setw( 3 ) << setfill( '0' ) << iter;
  string output_file = "data-" + ss.str() + ".h5";
  cerr << "Trial " << output_file << endl;
  ifstream output_name;
  output_name.open(output_file);
  while(output_name.is_open()){
    output_name.close();
    iter++;
    ss.str("");
    ss.clear();
    ss << setw( 3 ) << setfill( '0' ) << iter;
    output_file = "data-" + ss.str() + ".h5";
    cerr << "Trial " << output_file << endl;
    output_name.open(output_file);
  }



  double energy = 0, pressure = 0, hypervirial = 0;

  int count_list = 0, size_list = 1000;
  vector <float> list_pressure(size_list), list_energy(size_list), list_hypervirial(size_list);

  #if defined(SOFTCUDA)
  measure_e_p(energy, pressure, hypervirial);
  create(seed, (float)energy, (float)pressure, (float)hypervirial, output_file);
#endif
#if defined(HARDCUDA)
  double pressure_y = 0;
  //ecmc_measure_pressure(pressure);
  vector <float> list_pressure_y(size_list);
  measure_pressure(pressure, pressure_y);
  create(seed, 0, (float)pressure, (float)pressure_y, output_file);
#endif
  write_time(output_file,0,0);
 
  int count_output = 0;
  

  for(int MCS = 0; MCS < MaxMCS; MCS++){
    
    simulate(BETA);

    if(MCS % freq_energy == 0){
#if defined(SOFTCUDA)
      measure_e_p(energy, pressure, hypervirial);
      list_energy[count_list] = (float)energy;
      list_pressure[count_list] = (float)pressure;
      list_hypervirial[count_list] = (float)hypervirial;
      count_list++;
      if(count_list == size_list){
      	write_ep(list_energy, list_pressure, list_hypervirial, output_file, count_list);
      	count_list = 0;
      }
#elif defined(HARDCUDA)
      //ecmc_measure_pressure(pressure);
      measure_pressure(pressure, pressure_y);
      list_pressure[count_list] = (float)pressure;
      list_pressure_y[count_list] = (float)pressure_y;
      count_list++;
      if(count_list == size_list){
      	write_ep(list_energy, list_pressure, list_pressure_y, output_file, count_list);
      	count_list = 0;
      }
#endif
    }

    if(MCS % freq_output == 0){
      check_particle_number();
      dump_config(count_output, output_file, Lbox, rshiftx, rshifty);
      
      count_output++;
    }
  }
  
  check_particle_number();
  dump_config(count_output, output_file, Lbox, rshiftx, rshifty);
#if defined(SOFTCUDA)
  write_ep(list_energy, list_pressure, list_hypervirial, output_file, count_list);
#elif defined(HARDCUDA)
  write_ep(list_energy, list_pressure, list_pressure_y, output_file, count_list);
#endif
  write_time(output_file,mytime.cpu(), mytime.clock() );
  
  
  mytime.stats();
  cout<<"Run finished"<<endl;
  cerr<<"Run finished"<<endl;
  
  return 0;
}
